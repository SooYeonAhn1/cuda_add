#include "hip/hip_runtime.h"
// CUDA Vector Addition Example
// Based on:
// https://developer.nvidia.com/blog/even-easier-introduction-cuda/
// Modified for WSL2 + Unified Memory + Runtime Error Handling

#include <iostream>
#include <math.h>

#include <hip/hip_runtime.h>
#include <>

__global__
void add(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        y[i] = x[i] + y[i];
    }
}

int main() {
    int N = 1 << 20;
    float *x, *y;
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    for (int i = 0; i < N; ++i) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    int device = 0;
    // hipGetDeviceCount(&device);

    // if (device < 1) {
    //     std::cerr << "device ID: " << device << std::endl;
    // }

    device = 0;
    
    // hipSetDevice(device);
    hipGetDevice(&device);

    hipMemPrefetchAsync(x, N*sizeof(float), device);
    hipMemPrefetchAsync(y, N*sizeof(float), device);

    add<<<numBlocks, blockSize>>>(N, x, y);

    // hipError_t err = hipGetLastError();
    // if (err != hipSuccess) {
    //     std::cerr << "CUDA kernel launch failed: " << hipGetErrorString(err) << std::endl;
    // }

    hipDeviceSynchronize();

    float maxError = 0.0f;
    for (int i = 0; i < N; ++i) {
        maxError = fmax(maxError, fabs(3.0f - y[i]));
    }
    
    std::cout << "max error is: " << maxError << std::endl;

    hipFree(x);
    hipFree(y);

    return 0;
}
